#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>

#define ILP 8

__global__ void KernelMatrixAdd(int height, int width, float* A, float* B, float* result) {
	int tid = threadIdx.x + ILP * blockDim.x * blockIdx.x;
        for (int i = 0; i < ILP; ++i) {
                int current_tid = tid + i * blockDim.x;
		result[current_tid] = A[current_tid] + B[current_tid];
        }

}

