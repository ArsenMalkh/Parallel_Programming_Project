#include "hip/hip_runtime.h"
#include <MatrixMul.cuh>

#define Size 32

__global__ void MatrixMul(int heightA, int widthA, int widthB, float *matrixA, float *matrixB, float *matrixResult)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int idxA, idxB, Cval = 0;

    __shared__ float subA[Size][Size];
    __shared__ float subB[Size][Size];

    for (int x = 0; x < gridDim.x; x++)
    {
        idxA = row * widthA + x * blockDim.x + threadIdx.x;
        subA[threadIdx.y][threadIdx.x] = 0.0;
        if(idxA < heightA * widthA)
        {
            subA[threadIdx.y][threadIdx.x] = matrixA[idxA];
        }
        idxB = (x * blockDim.x + threadIdx.y) * widthB + col;
        subB[threadIdx.y][threadIdx.x] = 0.0;
        if(idxA < widthA * heightA && idxB < heightA * widthB)
        {
            subB[threadIdx.y][threadIdx.x] = matrixB[idxB];
        }
        __syncthreads();

        for (int k = 0; k < blockDim.y; k++)
        {
            Cval += subA[threadIdx.y][k] * subB[k][threadIdx.x];
        }
        __syncthreads();
    }
    if(row < heightA && col < widthB)
    {
        matrixResult[row * widthB + col] = Cval;
    }
}

