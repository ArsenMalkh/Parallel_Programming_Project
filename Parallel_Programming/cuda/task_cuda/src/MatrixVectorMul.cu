#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>

#define ILP 8

__global__ void MatrixVectorMul(int height, int width, float* matrix, float* vector, float* result) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	int col = blockIdx.y * blockDim.y + threadIdx.y;
	int stride_row = blockDim.x * gridDim.x;
	int stride_col = blockDim.y * gridDim.y;

	for(int i = row; i < height; i += stride_row) {
		for(int j = col; j< width; j += stride_col) {
			atomicAdd(&result[i], matrix[row * width + j] * vector[j]);
		}
	}

	/*for(int i = 0; i < ILP; ++i) {
		int current_x = row + i * blockDim.x;
		for(int j = 0;j < ILP; ++j) {
			int current_y = col + j * blockDim.y;
			atomicAdd(&result[current_x], matrix[current_x * width + current_y] * vector[current_y]);
		}
	}*/

}

