#include "hip/hip_runtime.h"
#include <ScalarMul.cuh>

/*
 * Calculates scalar multiplication for block
 */
__global__ void ScalarMulBlock(int numElements, float* vector1, float* vector2, float *result) {
	int index = blockDim.x * blockIdx.x + threadIdx.x;
	int stride = gridDim.x * blockDim.x;

	for(int i = index; i < numElements; i += stride) {
		atomicAdd(&result[blockIdx.x], vector1[i] * vector2[i]);
	}
}

