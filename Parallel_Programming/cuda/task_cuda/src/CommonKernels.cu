#include "hip/hip_runtime.h"
#include "CommonKernels.cuh"

#define ILP 8

__global__ void KernelMul(int numElements, float* vector1, float* vector2, float* result) {
        int tid = threadIdx.x + ILP * blockDim.x * blockIdx.x;
        for (int i = 0; i < ILP; ++i) {
                int current_tid = tid + i * blockDim.x;
                result[current_tid] = vector1[current_tid] * vector2[current_tid];
        }
}

