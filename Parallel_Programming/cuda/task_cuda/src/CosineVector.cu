#include "hip/hip_runtime.h"
#include <CosineVector.cuh>
#include <ScalarMulRunner.cuh>
#include <cmath>
#include <iostream>


float CosineVector(int numElements, float* vector1, float* vector2) {
	int blockSize = 1024;
	float firstAbs = sqrt(ScalarMulTwoReductions(numElements, vector1, vector1, blockSize));
	float secondAbs = sqrt(ScalarMulTwoReductions(numElements, vector2, vector2, blockSize));
	return ScalarMulTwoReductions(numElements, vector1, vector2, 1024) / (firstAbs * secondAbs);
}

