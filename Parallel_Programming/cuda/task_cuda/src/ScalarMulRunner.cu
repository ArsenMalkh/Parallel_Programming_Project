#include "hip/hip_runtime.h"
#include <ScalarMulRunner.cuh> 
#include <CommonKernels.cuh>
#include <ScalarMul.cuh>
#include <iostream>
 
#define ILP 8 

//Ivchenko code from Seminar
__global__ void Reduce(float* in_data, float* out_data) { extern __shared__
	float shared_data[];
 
    unsigned int tid = threadIdx.x; unsigned int index = blockIdx.x *
	    blockDim.x + threadIdx.x;
 
    shared_data[tid] = in_data[index]; __syncthreads();
 
    for (unsigned int s = 1; s < blockDim.x; s *= 2) { if (tid % (2 * s) == 0)
	    { shared_data[tid] += shared_data[tid + s]; } __syncthreads(); }
 
    if (tid == 0) { out_data[blockIdx.x] = shared_data[0]; } 
}
 
 
float ScalarMulTwoReductions(int numElements, float* vector1, float* vector2,
		int blockSize) {
        float *d_vec1, *d_vec2, *d_result;
        hipMalloc(&d_vec1, numElements * sizeof(float));
        hipMalloc(&d_vec2, numElements * sizeof(float));
        hipMalloc(&d_result, numElements * sizeof(float));
 
        hipMemcpy(d_vec1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_vec2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);
 
        int numblocks = (numElements + blockSize - 1) / blockSize;
	KernelMul<<<numblocks / ILP, blockSize>>>(numElements, d_vec1, d_vec2, d_result);
        hipDeviceSynchronize();
        float* d_res1;
        hipMalloc(&d_res1, numblocks * sizeof(float));
        Reduce<<<numblocks, blockSize, blockSize * sizeof(float)>>>(d_result, d_res1);
        hipDeviceSynchronize();
 
        float* d_out;
        hipMalloc(&d_out, sizeof(float));
 
        int reduceblockSize = (numblocks + blockSize - 1) / blockSize;
       	Reduce<<<reduceblockSize, blockSize, blockSize * sizeof(float)>>>(d_res1, d_out);
        hipDeviceSynchronize();
 
        float res;
        hipMemcpy(&res, d_out, sizeof(float), hipMemcpyDeviceToHost);
 
        hipFree(d_vec1);
        hipFree(d_vec2);
        hipFree(d_result);
        hipFree(d_out);
        hipFree(d_res1);
        return res;
 
 
}

float ScalarMulSumPlusReduction(int numElements, float* vector1, float* vector2, int blockSize) {
        float *d_vec1, *d_vec2, *d_result;
 
	int numblocks = (numElements + blockSize - 1) / blockSize;

        hipMalloc(&d_vec1, numElements * sizeof(float));
        hipMalloc(&d_vec2, numElements * sizeof(float));
        hipMalloc(&d_result, numblocks * sizeof(float));
 
        hipMemcpy(d_vec1, vector1, numElements * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_vec2, vector2, numElements * sizeof(float), hipMemcpyHostToDevice);
 
        ScalarMulBlock<<<numblocks, blockSize>>>(numElements, d_vec1, d_vec2, d_result);
        hipDeviceSynchronize();
 
        float* d_out;
        hipMalloc(&d_out, sizeof(float));
        int reduceblockSize = (numblocks + blockSize - 1) / blockSize;
        Reduce<<<reduceblockSize, numblocks, numblocks * sizeof(float)>>>(d_result, d_out);
        hipDeviceSynchronize();
        float res;
        hipMemcpy(&res, d_out, sizeof(float), hipMemcpyDeviceToHost);
 
        hipFree(d_vec1);
        hipFree(d_vec2);
        hipFree(d_result);
        hipFree(d_out);
        return res;
}
