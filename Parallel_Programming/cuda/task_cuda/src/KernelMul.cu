#include "hip/hip_runtime.h"
#include "KernelMul.cuh"

#define ILP 8

__global__ void KernelMul(int numElements, float* x, float* y, float* result) {
        int tid = threadIdx.x + ILP * blockDim.x * blockIdx.x;
        for (int i = 0; i < ILP; ++i) {
                int current_tid = tid + i * blockDim.x;
                result[current_tid] = x[current_tid] * y[current_tid];
        }
}
