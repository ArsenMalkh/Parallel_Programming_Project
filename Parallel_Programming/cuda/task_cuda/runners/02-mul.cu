#include <assert.h>
#include "KernelMul.cuh"
#include <iostream>
#include <fstream>

#define ILP 8

int main(int argc, char* argv[]) {
        
	int N = atoi(argv[1]);
        int blockSize = atoi(argv[2]);
        float *x = (float*)malloc(N * sizeof(float));
        float *y = (float*)malloc(N * sizeof(float));
        float *result = (float*)malloc(N * sizeof(float));

        float *d_x, *d_y, *d_result;

        hipMalloc(&d_x, N * sizeof(float));
        hipMalloc(&d_y, N * sizeof(float));
        hipMalloc(&d_result, N * sizeof(float));

        for (int i = 0; i < N; ++i) {
                x[i] = 3.0f;
                y[i] = 4.0f;
        }


        hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);
	
	hipEvent_t start;
        hipEvent_t stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

	int numBlocks = (N + blockSize - 1) / blockSize;
	hipEventRecord(start);
        KernelMul<<<numBlocks / ILP, blockSize>>>(N, d_x, d_y, d_result);
	hipEventRecord(stop);
	hipDeviceSynchronize();
        hipEventSynchronize(stop);

        float millis = 0;
        hipEventElapsedTime(&millis, start, stop);
        std::ofstream myfile;
        myfile.open("out.txt");
        myfile << blockSize<<" "<<N<<" "<<millis<<"\n";
        myfile.close();

        hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < N; ++i) {
                 assert(result[i] == 12.0f);
        }
        hipFree(d_x);
        hipFree(d_y);
        hipFree(d_result);
        free(x);
        free(y);
        free(result);
}
