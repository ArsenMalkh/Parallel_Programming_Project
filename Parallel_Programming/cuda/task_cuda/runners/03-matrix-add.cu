#include "hip/hip_runtime.h"
#include <KernelMatrixAdd.cuh>
#include <assert.h>
#include <fstream>
#include <cmath>

#define ILP 8

void FillMatrix(float* A, int width, int height, float x) {
	for(int row = 0; row < height; ++row) {
		for(int col = 0; col < width; ++col) {
			A[row * width + col] = x;	
		}
	}
}

int main(int argc, char* argv[]) {
	
	int N = atoi(argv[1]);
	int blockSize = atoi(argv[2]);
	
	int width = sqrt(N) / 2;
	int height = sqrt(N) * 2;
	int area = width * height;
	float *MA = (float*)malloc(area * sizeof(float));
	float *MB = (float*)malloc(area * sizeof(float));
	float *MC = (float*)malloc(area * sizeof(float));

	float *d_MA, *d_MB, *d_MC;
 	
	hipMalloc(&d_MA, area * sizeof(float));
        hipMalloc(&d_MB, area * sizeof(float));
        hipMalloc(&d_MC, area * sizeof(float));

	FillMatrix(MA, width, height, 3.0f);
	FillMatrix(MB, width, height, 4.0f);

	hipMemcpy(d_MA, MA, area * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_MB, MB, area * sizeof(float), hipMemcpyHostToDevice);

        int numBlocks = (area + blockSize - 1) / blockSize;
	
	hipEvent_t start;
        hipEvent_t stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
        KernelMatrixAdd<<<numBlocks / ILP, blockSize>>>(height, width, d_MA, d_MB, d_MC);
	hipEventRecord(stop);

        hipDeviceSynchronize();
        hipEventSynchronize(stop);

        float millis = 0;
        hipEventElapsedTime(&millis, start, stop);
        std::ofstream myfile;
        myfile.open ("out.txt");
        myfile << blockSize<<" "<<N<<" "<<millis<<"\n";
        myfile.close();


        hipMemcpy(MC, d_MC, area * sizeof(float), hipMemcpyDeviceToHost);

        for(int row = 0; row < height; ++row) {
		for(int col = 0; col < width; ++col) {
			assert(MC[row * width + col] == 7.0f);
		}
        }
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        free(MA);
        free(MB);
        free(MC);
}

