#include "hip/hip_runtime.h"
#include <MatrixVectorMul.cuh>
#include <assert.h>
#include <iostream>
#include <fstream>
#include <cmath>

#define ILP 8

void FillArray(float* A, int width, int height, float x) {
        for(int row = 0; row < height; ++row) {
                for(int col = 0; col < width; ++col) {
			A[row * width + col] = x;
			}
                }
        }

int main(int argc, char* argv[]) {
	int N = atoi(argv[1]);
        int Size = atoi(argv[2]);
	
	const int blockSizePoint = sqrt(Size); 
	int width = sqrt(N) / 2;
        int height = sqrt(N) * 2;
        int matrix_area = width * height;
	int vector_area = width * 1;
	int new_vector_area = height * 1;

        float *MA = (float*)malloc(matrix_area * sizeof(float));
        float *MB = (float*)malloc(vector_area * sizeof(float));
        float *MC = (float*)malloc(new_vector_area * sizeof(float));

        float *d_MA = NULL;
       	float *d_MB = NULL;
       	float *d_MC = NULL;
        hipMalloc(&d_MA, matrix_area * sizeof(float));
        hipMalloc(&d_MB, vector_area * sizeof(float));
        hipMalloc(&d_MC, new_vector_area * sizeof(float));

        FillArray(MA, width, height, 2.0f); //for Matrix
        FillArray(MB, 1,    width, 1.0f); // for Vector
        hipMemcpy(d_MA, MA, matrix_area * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_MB, MB, vector_area * sizeof(float), hipMemcpyHostToDevice);
        dim3 blockSize(blockSizePoint,blockSizePoint);
        dim3 numBlocks((height + blockSize.x - 1) / (blockSize.x), (width + blockSize.y - 1) / (blockSize.y));
        
	hipEvent_t start;
        hipEvent_t stop;

        hipEventCreate(&start);
        hipEventCreate(&stop);

        hipEventRecord(start);
	MatrixVectorMul<<<numBlocks, blockSize>>>(height, width, d_MA, d_MB, d_MC);
	hipEventRecord(stop);

        hipDeviceSynchronize();
        hipEventSynchronize(stop);

        float millis = 0;
        hipEventElapsedTime(&millis, start, stop);
        std::ofstream myfile;
        myfile.open ("out.txt");
        myfile << Size<<" "<<N<<" "<<millis<<"\n";
        myfile.close();


        hipMemcpy(MC, d_MC, new_vector_area * sizeof(float), hipMemcpyDeviceToHost);
	for (int row = 0; row <	height; ++row) {
		assert(MC[row] == 2.0f * width);
	}
        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        free(MA);
        free(MB);
       	free(MC);
}


