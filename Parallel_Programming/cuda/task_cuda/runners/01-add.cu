#include <assert.h>
#include "KernelAdd.cuh"
#include <iostream>
#include <fstream>

#define ILP 8

int main(int argc, char* argv[]) {

	int N = atoi(argv[1]);
	int blockSize = atoi(argv[2]);
	float *x = (float*)malloc(N * sizeof(float));
	float *y = (float*)malloc(N * sizeof(float));
    	float *result = (float*)malloc(N * sizeof(float));

	float *d_x, *d_y, *d_result;

	hipMalloc(&d_x, N * sizeof(float));
	hipMalloc(&d_y, N * sizeof(float));
    	hipMalloc(&d_result, N * sizeof(float));

	for (int i = 0; i < N; ++i) {
		x[i] = 1.0f;
		y[i] = 2.0f;
	}


	hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, N * sizeof(float), hipMemcpyHostToDevice);

	int numBlocks = (N + blockSize - 1) / blockSize;
	
	hipEvent_t start;
	hipEvent_t stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
	KernelAdd<<<numBlocks / ILP, blockSize>>>(N, d_x, d_y, d_result);
	hipEventRecord(stop);

	hipDeviceSynchronize();
  	hipEventSynchronize(stop);

	float millis = 0;
	hipEventElapsedTime(&millis, start, stop);
	std::ofstream myfile;
  	myfile.open ("out.txt");
	myfile << blockSize<<" "<<N<<" "<<millis<<"\n";
  	myfile.close();
	
	hipMemcpy(result, d_result, N * sizeof(float), hipMemcpyDeviceToHost);
	
	for(int i = 0; i < N; ++i) {
		 assert(result[i] == 3.0f);
	}
	hipFree(d_x);
	hipFree(d_y);
	hipFree(d_result);
	free(x);
	free(y);
	free(result);
}
