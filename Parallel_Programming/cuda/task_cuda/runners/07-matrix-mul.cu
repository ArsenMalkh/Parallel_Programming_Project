#include <MatrixMul.cuh>
#include <assert.h>

void FillArray(float* A, int width, int height, float x) {
        for(int row = 0; row < height; ++row) {
                for(int col = 0; col < width; ++col) {
                        A[row * width + col] = x;
                        }
                }
        }

void FillArray2(float* A, int width, int height, float x) {
        for(int row = 0; row < height; ++row) {
                for(int col = 0; col < width; ++col) {
                        if(row == col) {
                                A[row * width + col] = 2.0f;
                        }
                }
        }
}


int main() {
        int widthA = 1 << 10;
        int heightA = 1 << 10;
	int widthB = 1 << 10;
	int heightB = widthA; 
        int matrix_areaA = widthA * heightA;
	int matrix_areaB = widthB * heightB;

        float *MA = (float*)malloc(matrix_areaA * sizeof(float));
        float *MB = (float*)malloc(matrix_areaB * sizeof(float));
        float *MC = (float*)malloc(heightA * widthB * sizeof(float));

        float *d_MA = NULL;
        float *d_MB = NULL;
        float *d_MC = NULL;
        hipMalloc(&d_MA, matrix_areaA * sizeof(float));
        hipMalloc(&d_MB, matrix_areaB * sizeof(float));
        hipMalloc(&d_MC, heightA * widthB * sizeof(float));

        FillArray(MA, heightA, widthA, 2.25f);
        FillArray2(MB, heightB, widthB, 1.0f);
        hipMemcpy(d_MA, MA, matrix_areaA * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(d_MB, MB, matrix_areaB * sizeof(float), hipMemcpyHostToDevice);
        dim3 blockSize(32,32);
        dim3 numBlocks((heightA + blockSize.x - 1) / (blockSize.x), (widthB + blockSize.y - 1) / (blockSize.y));

        MatrixMul<<<numBlocks, blockSize>>>(heightA, widthA,  widthB, d_MA, d_MB, d_MC);

        hipDeviceSynchronize();
        hipMemcpy(MC, d_MC, heightA * widthB * sizeof(float), hipMemcpyDeviceToHost);

        for(int i = 0; i < heightA; ++i) {
                for(int j = 0; j < widthB; ++j) {
                        assert(MC[i*widthB + j] = 4.5f);
                }
        }

        hipFree(d_MA);
        hipFree(d_MB);
        hipFree(d_MC);
        free(MA);
        free(MB);
        free(MC);
}
               

